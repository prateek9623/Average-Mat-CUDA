#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "time.h";
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define RADIUS 1
#define MATRIX_SIZE 50
#define MAX 10

using namespace std;

 void fillRandom(int *matrix, int maxX, int maxY, int range, unsigned long seed) 
{
	 srand(seed);
	 for (int i = 0; i < maxX; i++)
		 for (int j = 0; j < maxY; j++)
			 *((matrix+i*maxY)+j) = rand() % MAX;
}

__global__ void findAverage(int *matrix, int *avgMatrix, int maxX, int maxY, int radius, int count, int sharedBlockSize) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = x + maxX * y;

	extern __shared__ int sharedData[];

	//int blockIndex = x + blockDim.x *y;
	//int blockSize = blockDim.x*blockDim.y;
	//int blockClipSize = sharedBlockSize / blockSize + 1;
	//int blockClipStart = blockClipSize * blockIndex;
	//if (blockClipStart < blockSize) {
	//	for (int i = 0; i < blockClipSize; i++) {
	//	}
	//}

	printf("\n%d %d", x, y);
}
//__global__ void findAverage(int *matrix, int *avgMatrix, int maxX, int maxY, int radius, int count) {
//	int x = threadIdx.x + blockIdx.x*blockDim.x;
//	int y = threadIdx.y + blockIdx.y*blockDim.y;
//	int index = x + maxX*y;
//	
//	if (x < maxX && y < maxY) {
//		int sum = 0;
//		int cou = 0;
//		for (int offsetY = y - radius; offsetY <= y + radius && offsetY < maxY; offsetY++) {
//			for (int offsetX = x - radius; offsetX <= x + radius && offsetX < maxX; offsetX++) {
//				if (offsetX >= 0 && offsetY >= 0)
//				{
//					int indexOffset = offsetY * maxX + offsetX;
//					sum += matrix[indexOffset];
//				}
//			}
//		}
//		avgMatrix[index] = sum / count;
//		//__syncthreads();
//		//printf("%d ", avgMatrix[index]);
//	}
//}


int main()
{
	int matrix[MATRIX_SIZE][MATRIX_SIZE];
	int avgMatrix[MATRIX_SIZE][MATRIX_SIZE];

	int *dMatrix;
	int *dAvgMatrix;

	hipFree(0);

	fillRandom((int*)matrix, MATRIX_SIZE, MATRIX_SIZE, 10, time(NULL));
	int totalElements = MATRIX_SIZE * MATRIX_SIZE;

	if (hipMalloc(&dMatrix, sizeof(int)*totalElements) != hipSuccess) {
		cerr << "Couldn't allocate memory for matrix";
		hipFree(dMatrix);
	};

	if (hipMalloc(&dAvgMatrix, sizeof(int)*totalElements) != hipSuccess) {
		cerr << "Couldn't allocate memory for Average Matrix";
		hipFree(dAvgMatrix);
	};

	if (hipMemcpy(dMatrix, matrix, sizeof(int)*totalElements, hipMemcpyHostToDevice) != hipSuccess) {
		cerr << "Couldn,t initialiZe device Original Matrix";
		hipFree(dMatrix);
		hipFree(dAvgMatrix);
	}

	if (hipMemset(dAvgMatrix, 0, sizeof(int)*totalElements) != hipSuccess) {
		cerr << "Couldn,t initialiZe device Average Matrix";
		hipFree(dMatrix);
		hipFree(dAvgMatrix);
	}

	const dim3 blockSize(4, 4);
	const dim3 gridSize((MATRIX_SIZE + blockSize.x - 1) / blockSize.x, (MATRIX_SIZE + blockSize.y - 1) / blockSize.y);
	int count = (RADIUS * 2 + 1)*(RADIUS * 2 + 1);

	int sharedMemSpace = (blockSize.x+2*RADIUS)*(blockSize.y+2*RADIUS) ;
	
	findAverage <<<gridSize, blockSize, sharedMemSpace * sizeof(int) >>> (dMatrix, dAvgMatrix, MATRIX_SIZE, MATRIX_SIZE, RADIUS, count, sharedMemSpace);

	hipDeviceSynchronize();

	if (hipGetLastError() != hipSuccess) {
		cerr << "kernel launch failed: " << hipGetErrorString(hipGetLastError());
		hipFree(dMatrix);
		hipFree(dAvgMatrix);
		exit(1);
	}

	if (hipMemcpy(avgMatrix, dAvgMatrix, sizeof(int)*totalElements, hipMemcpyDeviceToHost) != hipSuccess) {
		cerr << "Couldn't copy original matrix memory from device to host";
		hipFree(dMatrix);
		hipFree(dAvgMatrix);
		exit(1);
	}

	cout << endl << endl;
	for (int i = 0; i < MATRIX_SIZE; i++) {
		for (int j = 0; j < MATRIX_SIZE; j++) {
			cout << matrix[i][j] << " ";
		}
		cout << endl;
	}

	cout << endl<<endl;
	for (int i = 0; i < MATRIX_SIZE; i++){
		for (int j = 0; j < MATRIX_SIZE; j++) {
			cout << avgMatrix[i][j] << " ";
		}
		cout << endl;
	}


    return 0;
}
